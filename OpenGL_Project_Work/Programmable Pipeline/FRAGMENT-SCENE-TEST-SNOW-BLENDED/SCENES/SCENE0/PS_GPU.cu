
#include <hip/hip_runtime.h>

__global__ void sinwave_vbo_kernel(float4 *pos, float3 *posDir, float3 *posDirInitial, float3 *posGravity, float *posLife, float *posFade, unsigned int width, unsigned int height, float velocity, float animTime)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	long index = y * width + x;

	pos[index].x += (posDir[index].x / (velocity * 1000))*4;
	pos[index].y += (posDir[index].y / (velocity * 1000))*2;
	pos[index].z += (posDir[index].z / (velocity * 1000))*1;
	pos[index].w = 1.0;
	//sin(4.0 * posDir[index].z * animTime) * cos(4.0 * posDir[index].y * animTime) * 0.5f;
	posDir[index].x += posGravity[index].x;
	posDir[index].y += posGravity[index].y;
	posDir[index].z += posGravity[index].z;
	
//	pos[index].x < bx && pos[index].y < by && pos[index].z < bz // compare bonding box and end then end up life
	
	posLife[index] -= posFade[index];

	/*if(pos[index].z < 0.0)
	{
		pos[index].z = 0.0;
	}*/

	if (posLife[index] < -30.0 /*pos[index].y <= -50.0*/) 
	{
		posLife[index] = 2.0;
		pos[index] = make_float4(0.0, 0.0, 0.0, 1.0);
		posDir[index] = posDirInitial[index];
		/*int co1 = (int) (pos[index].x / 50.0);
		float rem1 = pos[index].x - (co1  * 50.0);

		int co2 = (int) (pos[index].y / 50.0);
		float rem2 = pos[index].y - (co2 * 50.0);

		int co3 = (int)(pos[index].z / 50.0);
		float rem3 = pos[index].z - (co3 * 50.0);
		
		posDir[index].x = (float)((rem1) - 26.0)*10.0;
		posDir[index].y = (float)((rem2) - 25.0)*10.0;
		posDir[index].z = (float)((rem3) - 25.0)*10.0;*/
	}

	//pos[y*width+x] = make_float4(u,w,v,1.0);
	
	return;
}

//Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b)
{
	return (a + (b - 1)) / b;
}

void launchCudaKernel(float4* pos, float3 *posDir, float3 *posDirInitial, float3 *posGravity, float *posLife, float *posFade, unsigned int width, unsigned int height, float velocity, float animTime)
{
	dim3 block(8, 8, 1);
	dim3 grid(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1.0);
	sinwave_vbo_kernel <<< grid, block >>> (pos, posDir, posDirInitial, posGravity, posLife, posFade, width, height, velocity, animTime);
}
